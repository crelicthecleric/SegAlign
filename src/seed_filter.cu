#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/binary_search.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/scan.h>
#include <thrust/unique.h>
#include "parameters.h"
#include "seed_filter.h"

// Each segment is 16B
// With 64MB for the HSPs array per 1GB GPU memory
// With higher GPU memory, the size just linearly increases

#define MAX_HITS_PER_GB 4194304

// Control Variables
std::mutex mu;
std::condition_variable cv;
std::vector<int> available_gpus;

int NUM_DEVICES;
int MAX_SEEDS;
int MAX_HITS;

uint32_t seed_size;
int **d_sub_mat;
int xdrop;
int hspthresh;
bool noentropy;

char** d_seq;
char** d_seq_rc;
uint32_t seq_len;

uint32_t** d_index_table;
uint32_t** d_pos_table;

uint64_t** d_seed_offsets;

uint32_t** d_hit_num_array;
std::vector<thrust::device_vector<uint32_t> > d_hit_num_vec;

uint32_t** d_done;
std::vector<thrust::device_vector<uint32_t> > d_done_vec;

segment** d_hsp;
std::vector<thrust::device_vector<segment> > d_hsp_vec;

segment** d_hsp_reduced;
std::vector<thrust::device_vector<segment> > d_hsp_reduced_vec;

// wrap of hipSetDevice error checking in one place.  
static inline void check_cuda_setDevice(int device_id, const char* tag) {
    hipError_t err = hipSetDevice(device_id);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipSetDevice failed for device %d in %s failed with error \" %s \" \n", device_id, tag, hipGetErrorString(err));
        exit(11);
    }
}

// wrap of hipMalloc error checking in one place.  
static inline void check_cuda_malloc(void** buf, size_t bytes, const char* tag) {
    hipError_t err = hipMalloc(buf, bytes);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMalloc of %lu bytes for %s failed with error \" %s \" \n", bytes, tag, hipGetErrorString(err));
        exit(12);
    }
}
	 
// wrap of hipMemcpy error checking in one place.  
static inline void check_cuda_memcpy(void* dst_buf, void* src_buf, size_t bytes, hipMemcpyKind kind, const char* tag) {
    hipError_t err = hipMemcpy(dst_buf, src_buf, bytes, kind);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipMemcpy of %lu bytes for %s failed with error \" %s \" \n", bytes, tag, hipGetErrorString(err));
        exit(13);
    }
}
	 
// wrap of hipFree error checking in one place.  
static inline void check_cuda_free(void* buf, const char* tag) {
    hipError_t err = hipFree(buf);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: hipFree for %s failed with error \" %s \" \n", tag, hipGetErrorString(err));
        exit(14);
    }
}
	 
struct hspDiagEqual{
    __host__ __device__
        bool operator()(segment x, segment y){
        return ( ( (x.ref_start - x.query_start) == (y.ref_start - y.query_start) ) &&  ( ( (x.ref_start >= y.ref_start) && ( (x.ref_start + x.len) <= (y.ref_start + y.len) )  ) || ( ( y.ref_start >= x.ref_start ) && ( (y.ref_start + y.len) <= (x.ref_start + x.len) ) ) ) );
    }
};

struct hspDiagComp{
    __host__ __device__
        bool operator()(segment x, segment y){
            if((x.ref_start - x.query_start) < (y.ref_start - y.query_start))
                return true;
            else if((x.ref_start - x.query_start) == (y.ref_start - y.query_start)){
                if(x.ref_start < y.ref_start)
                    return true;
                else if(x.ref_start == y.ref_start){
                    if(x.query_start < y.query_start)
                        return true;
                    else if(x.query_start == y.query_start){
                        if(x.score > y.score)
                            return true;
                        else
                            return false;
                    }
                    else
                        return false;
                }
                else
                    return false;
            }
            else 
                return false;
    }
};

struct hspEqual{
    __host__ __device__
        bool operator()(segment x, segment y){
        return ((x.ref_start == y.ref_start) && (x.query_start == y.query_start) && (x.len == y.len) && (x.score == y.score));
    }
};

struct hspFinalComp{
    __host__ __device__
        bool operator()(segment x, segment y){
            if(x.query_start < y.query_start)
                return true;
            else if(x.query_start == y.query_start){
                if(x.score > y.score)
                    return true;
                else if(x.score == y.score){
                    if(x.ref_start > y.ref_start)
                        return true;
                    else
                        return false;
                }
                else
                    return false;
            }
            else
                return false;
    }
};

struct hspComp{
    __host__ __device__
        bool operator()(segment x, segment y){
            if(x.query_start < y.query_start)
                return true;
            else if(x.query_start == y.query_start){
                if(x.len > y.len)
                    return true;
                else if(x.len == y.len){
                    if(x.ref_start < y.ref_start)
                        return true;
                    else if(x.ref_start == y.ref_start){
                        if(x.score > y.score)
                            return true;
                        else
                            return false;
                    }
                    else
                        return false;
                }
                else
                    return false;
            }
            else 
                return false;
    }
};

__global__
void compress_string (uint32_t len, char* src_seq, char* dst_seq){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = X_NT;
        if (ch == 'A')
            dst = A_NT;
        else if (ch == 'C')
            dst = C_NT;
        else if (ch == 'G')
            dst = G_NT;
        else if (ch == 'T')
            dst = T_NT;
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't'))
            dst = L_NT;
        else if ((ch == 'n') || (ch == 'N'))
            dst = N_NT;
        else if (ch == '&')
            dst = E_NT;
        dst_seq[i] = dst;
    }
}

__global__
void compress_string_rev_comp (uint32_t len, char* src_seq, char* dst_seq, char* dst_seq_rc){ 
    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    for (uint32_t i = start; i < len; i += stride) {
        char ch = src_seq[i];
        char dst = X_NT;
        char dst_rc = X_NT;
        if (ch == 'A'){
            dst = A_NT;
            dst_rc = T_NT;
        }
        else if (ch == 'C'){ 
            dst = C_NT;
            dst_rc = G_NT;
        }
        else if (ch == 'G'){
            dst = G_NT;
            dst_rc = C_NT;
        }
        else if (ch == 'T'){
            dst = T_NT;
            dst_rc = A_NT;
        }
        else if ((ch == 'a') || (ch == 'c') || (ch == 'g') || (ch == 't')){
            dst = L_NT;
            dst_rc = L_NT;
        }
        else if ((ch == 'n') || (ch == 'N')){
            dst = N_NT;
            dst_rc = N_NT;
        }
        else if (ch == '&'){
            dst = E_NT;
            dst_rc = E_NT;
        }
        dst_seq[i] = dst;
        dst_seq_rc[len -1 -i] = dst_rc;
    }
}

__global__
void find_num_hits (int num_seeds, const uint32_t* __restrict__ d_index_table, uint64_t* seed_offsets, uint32_t* seed_hit_num){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;

    uint32_t num_seed_hit;
    uint32_t seed;
    
    for (uint32_t id = start; id < num_seeds; id += stride) {
        seed = (seed_offsets[id] >> 32);

        // start and end from the seed block_id table
        num_seed_hit = d_index_table[seed];
        if (seed > 0){
            num_seed_hit -= d_index_table[seed-1];
        }

        seed_hit_num[id] = num_seed_hit;
    }
}

__global__
void find_hits (const uint32_t* __restrict__  d_index_table, const uint32_t* __restrict__ d_pos_table, uint64_t*  d_seed_offsets, uint32_t seed_size, uint32_t* seed_hit_num, int num_hits, segment* d_hsp, uint32_t start_seed_index, uint32_t start_hit_index, uint32_t ref_start, uint32_t ref_end){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t start, end;
    __shared__ uint32_t seed;
    __shared__ uint64_t seed_offset;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc;
    __shared__ uint32_t seed_hit_prefix;

    if(thread_id == 0){
        seed_offset = d_seed_offsets[block_id+start_seed_index];
        seed = (seed_offset >> 32);
        query_loc = ((seed_offset << 32) >> 32) + seed_size;

        // start and end from the seed block_id table
        end = d_index_table[seed];
        start = 0;
        if (seed > 0){
            start = d_index_table[seed-1];
        }
        seed_hit_prefix = seed_hit_num[block_id+start_seed_index]; 
    }
    __syncthreads();


    for (int id1 = start; id1 < end; id1 += NUM_WARPS) {
        if(id1+warp_id < end){ 
            if(lane_id == 0){ 
                ref_loc[warp_id]   = d_pos_table[id1+warp_id] + seed_size;
                int dram_address = seed_hit_prefix -id1 - warp_id+start-1-start_hit_index;

                d_hsp[dram_address].ref_start = ref_loc[warp_id];
                d_hsp[dram_address].query_start = query_loc; 
                d_hsp[dram_address].len = 0;
                if(ref_loc[warp_id] >= ref_start && ref_loc[warp_id] <= ref_end){
                    d_hsp[dram_address].score = 0;
                }
                else{
                    d_hsp[dram_address].score = -1;
                }
            }
        }
    }
}

__global__
void find_hsps (const char* __restrict__  d_ref_seq, const char* __restrict__  d_query_seq, uint32_t ref_len, uint32_t query_len, int *d_sub_mat, bool noentropy, int xdrop, int hspthresh, int num_hits, segment* d_hsp, uint32_t* d_done){

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int num_blocks = gridDim.x;
    int warp_size = warpSize;
    int lane_id = thread_id%warp_size;
    int warp_id = (thread_id-lane_id)/warp_size;

    __shared__ uint32_t ref_loc[NUM_WARPS];
    __shared__ uint32_t query_loc[NUM_WARPS];
    __shared__ int total_score[NUM_WARPS];
    __shared__ int prev_score[NUM_WARPS];
    __shared__ int prev_max_score[NUM_WARPS];
    __shared__ int prev_max_pos[NUM_WARPS];
    __shared__ bool find_hsp[NUM_WARPS]; 
    __shared__ bool edge_found[NUM_WARPS]; 
    __shared__ bool xdrop_found[NUM_WARPS]; 
    __shared__ bool new_max_found[NUM_WARPS]; 
    __shared__ uint32_t left_extent[NUM_WARPS];
    __shared__ int extent[NUM_WARPS];
    __shared__ uint32_t tile[NUM_WARPS];
    __shared__ double entropy[NUM_WARPS];

    int thread_score;
    int max_thread_score;
    int max_pos;
    int temp_pos;
    bool xdrop_done;
    bool temp_xdrop_done;
    int temp;
    short count[4];
    short count_del[4];
    char r_chr;
    char q_chr;
    uint32_t ref_pos;
    uint32_t query_pos;
    int pos_offset;

    __shared__ int sub_mat[NUC2];

    if(thread_id < NUC2){
        sub_mat[thread_id] = d_sub_mat[thread_id];
    }
    __syncthreads();

    for(int hid0 = block_id*NUM_WARPS; hid0 < num_hits; hid0 += NUM_WARPS*num_blocks){ 
        int hid = hid0 + warp_id; 

        if(hid < num_hits){
            if(lane_id == 0){
                ref_loc[warp_id] = d_hsp[hid].ref_start;
                query_loc[warp_id] = d_hsp[hid].query_start;
                total_score[warp_id] = 0; 
                if(d_hsp[hid].score < 0){
                    find_hsp = false;
                }
                else{
                    find_hsp = true;
                }
            }
        }
        else{
            if(lane_id == 0){

                ref_loc[warp_id] = d_hsp[hid0].ref_start;
                query_loc[warp_id] = d_hsp[hid0].query_start;
                total_score[warp_id] = 0; 
            }
        }
        __syncwarp();


        //////////////////////////////////////////////////////////////////
        //Right extension

        if(lane_id ==0){
            if(find_hsp){
                edge_found[warp_id] = false;
            }
            else{
                edge_found[warp_id] = true;
            }
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            new_max_found[warp_id] = false;
            entropy[warp_id] = 1.0f;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = -1;
            extent[warp_id] = 0;
        }

        count[0] = 0;
        count[1] = 0;
        count[2] = 0;
        count[3] = 0;
        count_del[0] = 0;
        count_del[1] = 0;
        count_del[2] = 0;
        count_del[3] = 0;
        max_pos = 0;

        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id + tile[warp_id];
            ref_pos   = ref_loc[warp_id] + pos_offset;
            query_pos = query_loc[warp_id] + pos_offset;
            thread_score = 0;

            if(ref_pos < ref_len && query_pos < query_len){
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }


            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }

            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp_xdrop_done = __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);

                if(lane_id >= offset){
                    xdrop_done |= temp_xdrop_done;
                }
            }

            if(xdrop_done == 1){
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1){

                if(max_pos > prev_max_pos[warp_id])
                    new_max_found[warp_id] = true;
                else
                    new_max_found[warp_id] = false;

                if(xdrop_done){
                    total_score[warp_id] += max_thread_score;
                    xdrop_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_pos >= ref_len || query_pos >= query_len){
                    total_score[warp_id] += max_thread_score;
                    edge_found[warp_id] = true;
                    extent[warp_id] = max_pos;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+= warp_size;
                }
            }
            __syncwarp();

            if(new_max_found[warp_id]){
                for(int i = 0; i < 4; i++){
                    count[i] = count[i] + count_del[i];
                    count_del[i] = 0;
                }
            }
            __syncwarp();

            if(r_chr == q_chr){
                if(pos_offset <= prev_max_pos[warp_id]){
                    count[r_chr] += 1;
                }
                else{
                    count_del[r_chr] += 1;
                }
            }
            __syncwarp();

        }

        __syncwarp();

        ////////////////////////////////////////////////////////////////
        //Left extension

        if(lane_id ==0){
            if(find_hsp){
                edge_found[warp_id] = false;
            }
            else{
                edge_found[warp_id] = true;
            }
            tile[warp_id] = 0;
            xdrop_found[warp_id] = false;
            new_max_found[warp_id] = false;
            prev_score[warp_id] = 0;
            prev_max_score[warp_id] = 0;
            prev_max_pos[warp_id] = 0;
            left_extent[warp_id] = 0;
        }

        count_del[0] = 0;
        count_del[1] = 0;
        count_del[2] = 0;
        count_del[3] = 0;
        max_pos = 0;
        __syncwarp();

        while(!xdrop_found[warp_id] && !edge_found[warp_id]){
            pos_offset = lane_id+1+tile[warp_id];
            thread_score = 0;

            if(ref_loc[warp_id] >= pos_offset  && query_loc[warp_id] >= pos_offset){
                ref_pos   = ref_loc[warp_id] - pos_offset;
                query_pos = query_loc[warp_id] - pos_offset;
                r_chr = d_ref_seq[ref_pos];
                q_chr = d_query_seq[query_pos];
                thread_score = sub_mat[r_chr*NUC+q_chr];
            }

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, thread_score, offset);

                if(lane_id >= offset){
                    thread_score += temp;
                }
            }

            thread_score += prev_score[warp_id];
            if(thread_score > prev_max_score[warp_id]){
                max_thread_score = thread_score;
                max_pos = pos_offset;
            }
            else{
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }

            xdrop_done = ((max_thread_score-thread_score) > xdrop);
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp_xdrop_done = __shfl_up_sync(0xFFFFFFFF, xdrop_done, offset);

                if(lane_id >= offset){
                    xdrop_done |= temp_xdrop_done;
                }
            }

            if(xdrop_done == 1){
                max_thread_score = prev_max_score[warp_id];
                max_pos = prev_max_pos[warp_id];
            }
            __syncwarp();

#pragma unroll
            for (int offset = 1; offset < warp_size; offset = offset << 1){
                temp = __shfl_up_sync(0xFFFFFFFF, max_thread_score, offset);
                temp_pos = __shfl_up_sync(0xFFFFFFFF, max_pos, offset);

                if(lane_id >= offset){
                    if(temp >= max_thread_score){
                        max_thread_score = temp;
                        max_pos = temp_pos;
                    }
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1){

                if(max_pos > prev_max_pos[warp_id])
                    new_max_found[warp_id] = true;
                else
                    new_max_found[warp_id] = false;

                if(xdrop_done){
                    total_score[warp_id]+=max_thread_score;
                    xdrop_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else if(ref_loc[warp_id] < pos_offset || query_loc[warp_id] < pos_offset){
                    total_score[warp_id]+=max_thread_score;
                    edge_found[warp_id] = true;
                    left_extent[warp_id] = max_pos;
                    extent[warp_id] += left_extent[warp_id];
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id] = max_pos;
                }
                else{
                    prev_score[warp_id] = thread_score;
                    prev_max_score[warp_id] = max_thread_score;
                    prev_max_pos[warp_id] = max_pos;
                    tile[warp_id]+=warp_size;
                }
            }
            __syncwarp();

            if(new_max_found[warp_id]){
                for(int i = 0; i < 4; i++){
                    count[i] = count[i] + count_del[i];
                    count_del[i] = 0;
                }
            }
            __syncwarp();

            if(r_chr == q_chr){
                if(pos_offset <= prev_max_pos[warp_id]){
                    count[r_chr] += 1;
                }
                else{
                    count_del[r_chr] += 1;
                }
            }
            __syncwarp();

        }

        //////////////////////////////////////////////////////////////////

        if(total_score[warp_id] >= hspthresh && total_score[warp_id] <= 3*hspthresh && !noentropy){
            for(int i = 0; i < 4; i++){
#pragma unroll
                for (int offset = 1; offset < warp_size; offset = offset << 1){
                    count[i] += __shfl_up_sync(0xFFFFFFFF, count[i], offset);
                }
            }
            __syncwarp();

            if(lane_id == warp_size-1 && ((count[0]+count[1]+count[2]+count[3]) >= 20)){

                entropy[warp_id] = 0.f;
                for(int i = 0; i < 4; i++){
                    entropy[warp_id] += ((double) count[i])/((double) (extent[warp_id]+1)) * ((count[i] != 0) ? log(((double) count[i]) / ((double) (extent[warp_id]+1))): 0.f); 
                }
                entropy[warp_id] = -entropy[warp_id]/log(4.0f);
            }
        }
        __syncwarp();

        //////////////////////////////////////////////////////////////////

        if(hid < num_hits){
            if(lane_id == 0){

                if( ((int) (((float) total_score[warp_id])  * entropy[warp_id])) >= hspthresh){
                    d_hsp[hid].ref_start = ref_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id] - left_extent[warp_id];
                    d_hsp[hid].len = extent[warp_id];
                    if(entropy[warp_id] > 0)
                        d_hsp[hid].score = total_score[warp_id]*entropy[warp_id];
                    d_done[hid] = 1;
                }
                else{
                    d_hsp[hid].ref_start = ref_loc[warp_id];
                    d_hsp[hid].query_start = query_loc[warp_id];
                    d_hsp[hid].len = 0;
                    d_hsp[hid].score = 0;
                    d_done[hid] = 0;
                }
            }
        }
        __syncwarp();
    }
}

__global__
void compress_output (uint32_t* d_done, segment* d_hsp, segment* d_hsp_reduced, int num_hits){

    int thread_id = threadIdx.x;
    int block_dim = blockDim.x;
    int grid_dim = gridDim.x;
    int block_id = blockIdx.x;

    int stride = block_dim * grid_dim;
    uint32_t start = block_dim * block_id + thread_id;
    int index = 0;

    for (uint32_t id = start; id < num_hits; id += stride) {
        index = d_done[id];

        if(id > 0){
            if(index > d_done[id-1]){
                d_hsp_reduced[index-1] = d_hsp[id];
            }
        }
        else{
            if(index == 1){
                d_hsp_reduced[0] = d_hsp[0];
            }
        }
    }
}

std::vector<segment> SeedAndFilter (std::vector<uint64_t> seed_offset_vector, bool rev, uint32_t ref_start, uint32_t ref_end){

    uint32_t num_hits = 0;
    uint32_t total_anchors = 0;

    uint32_t num_seeds = seed_offset_vector.size();
    assert(num_seeds <= MAX_SEEDS);

    uint64_t* tmp_offset = (uint64_t*) malloc(num_seeds*sizeof(uint64_t));
    for (uint32_t i = 0; i < num_seeds; i++) {
        tmp_offset[i] = seed_offset_vector[i];
    }

    int g;
    std::unique_lock<std::mutex> locker(mu);
    if (available_gpus.empty()) {
        cv.wait(locker, [](){return !available_gpus.empty();});
    }
    g = available_gpus.back();
    available_gpus.pop_back();
    locker.unlock();

    check_cuda_setDevice(g, "SeedAndFilter");

    check_cuda_memcpy((void*)d_seed_offsets[g], (void*)tmp_offset, num_seeds*sizeof(uint64_t), hipMemcpyHostToDevice, "seed_offsets");

    find_num_hits <<<MAX_BLOCKS, MAX_THREADS>>> (num_seeds, d_index_table[g], d_seed_offsets[g], d_hit_num_array[g]);

    thrust::inclusive_scan(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin() + num_seeds, d_hit_num_vec[g].begin());

    check_cuda_memcpy((void*)&num_hits, (void*)(d_hit_num_array[g]+num_seeds-1), sizeof(uint32_t), hipMemcpyDeviceToHost, "num_hits");
    
    int num_iter = num_hits/MAX_HITS+1;
    uint32_t iter_hit_limit = MAX_HITS;
    thrust::device_vector<uint32_t> limit_pos (num_iter); 

    for(int i = 0; i < num_iter-1; i++){
        thrust::device_vector<uint32_t>::iterator result_end = thrust::lower_bound(d_hit_num_vec[g].begin(), d_hit_num_vec[g].begin()+num_seeds, iter_hit_limit);
        uint32_t pos = thrust::distance(d_hit_num_vec[g].begin(), result_end)-1;
        iter_hit_limit = d_hit_num_vec[g][pos]+MAX_HITS;
        limit_pos[i] = pos;
    }

    limit_pos[num_iter-1] = num_seeds-1;

    segment** h_hsp = (segment**) malloc(num_iter*sizeof(segment*));
    uint32_t* num_anchors = (uint32_t*) calloc(num_iter, sizeof(uint32_t));

    uint32_t start_seed_index = 0;
    uint32_t start_hit_val = 0;
    uint32_t iter_num_seeds, iter_num_hits;

    if(num_hits > 0){
        
        for(int i = 0; i < num_iter; i++){
            iter_num_seeds = limit_pos[i] + 1 - start_seed_index;
            iter_num_hits  = d_hit_num_vec[g][limit_pos[i]] - start_hit_val;

            find_hits <<<iter_num_seeds, BLOCK_SIZE>>> (d_index_table[g], d_pos_table[g], d_seed_offsets[g], seed_size, d_hit_num_array[g], iter_num_hits, d_hsp[g], start_seed_index, start_hit_val, ref_start, ref_end);

            if(rev){
                find_hsps <<<1024, BLOCK_SIZE>>> (d_seq[g], d_seq_rc[g], seq_len, seq_len, d_sub_mat[g], noentropy, xdrop, hspthresh, iter_num_hits, d_hsp[g], d_done[g]);
            }
            else{
                find_hsps <<<1024, BLOCK_SIZE>>> (d_seq[g], d_seq[g], seq_len, seq_len, d_sub_mat[g], noentropy, xdrop, hspthresh, iter_num_hits, d_hsp[g], d_done[g]);
            }

            thrust::inclusive_scan(d_done_vec[g].begin(), d_done_vec[g].begin() + iter_num_hits, d_done_vec[g].begin());

            check_cuda_memcpy((void*)&num_anchors[i], (void*)(d_done[g]+iter_num_hits-1), sizeof(uint32_t), hipMemcpyDeviceToHost, "num_anchors");

            if(num_anchors[i] > 0){
                compress_output <<<MAX_BLOCKS, MAX_THREADS>>>(d_done[g], d_hsp[g], d_hsp_reduced[g], iter_num_hits);

                thrust::stable_sort(d_hsp_reduced_vec[g].begin(), d_hsp_reduced_vec[g].begin()+num_anchors[i], hspComp());
                
                thrust::device_vector<segment>::iterator result_end = thrust::unique_copy(d_hsp_reduced_vec[g].begin(), d_hsp_reduced_vec[g].begin()+num_anchors[i], d_hsp_vec[g].begin(),  hspEqual());

                num_anchors[i] = thrust::distance(d_hsp_vec[g].begin(), result_end), num_anchors[i];

                thrust::stable_sort(d_hsp_vec[g].begin(), d_hsp_vec[g].begin()+num_anchors[i], hspDiagComp());
                
                thrust::device_vector<segment>::iterator result_end2 = thrust::unique_copy(d_hsp_vec[g].begin(), d_hsp_vec[g].begin()+num_anchors[i], d_hsp_reduced_vec[g].begin(),  hspDiagEqual());

                num_anchors[i] = thrust::distance(d_hsp_reduced_vec[g].begin(), result_end2), num_anchors[i];

                thrust::stable_sort(d_hsp_reduced_vec[g].begin(), d_hsp_reduced_vec[g].begin()+num_anchors[i], hspFinalComp());

                total_anchors += num_anchors[i];

                h_hsp[i] = (segment*) calloc(num_anchors[i], sizeof(segment));

                check_cuda_memcpy((void*)h_hsp[i], (void*)d_hsp_reduced[g], num_anchors[i]*sizeof(segment), hipMemcpyDeviceToHost, "hsp_output");
            }

            start_seed_index = limit_pos[i] + 1;
            start_hit_val = d_hit_num_vec[g][limit_pos[i]];
        }
    }

    limit_pos.clear();

    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(g);
        locker.unlock();
        cv.notify_one();
    }
    std::vector<segment> gpu_filter_output;

    segment first_el;
    first_el.len = total_anchors;
    first_el.score = num_hits;
    gpu_filter_output.push_back(first_el);

    if(total_anchors > 0){
        for(int it = 0; it < num_iter; it++){

            for(int i = 0; i < num_anchors[it]; i++){
                gpu_filter_output.push_back(h_hsp[it][i]);
            }
        }
        free(h_hsp);
    }
    
    free(tmp_offset);
    return gpu_filter_output;
}

int InitializeProcessor (int num_gpu, bool transition, uint32_t WGA_CHUNK, uint32_t input_seed_size, int* sub_mat, int input_xdrop, int input_hspthresh, bool input_noentropy){

    int nDevices;

    hipError_t err = hipGetDeviceCount(&nDevices);
    if (err != hipSuccess) {
        fprintf(stderr, "Error: No GPU device found!\n");
        exit(1);
    }

    if(num_gpu == -1){
        NUM_DEVICES = nDevices; 
    }
    else{
        if(num_gpu <= nDevices){
            NUM_DEVICES = num_gpu;
        }
        else{
            fprintf(stderr, "Requested GPUs greater than available GPUs\n");
            exit(10);
        }
    }

    fprintf(stderr, "Using %d GPU(s)\n", NUM_DEVICES);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    float global_mem_gb = static_cast<float>(deviceProp.totalGlobalMem / 1073741824.0f);

    if(transition)
        MAX_SEEDS = 13*WGA_CHUNK;
    else
        MAX_SEEDS = WGA_CHUNK;

    MAX_HITS = MAX_HITS_PER_GB*global_mem_gb;

    seed_size = input_seed_size;
    xdrop = input_xdrop;
    hspthresh = input_hspthresh;
    noentropy = input_noentropy;

    d_sub_mat = (int**) malloc(NUM_DEVICES*sizeof(int*));

    d_seq = (char**) malloc(NUM_DEVICES*sizeof(char*));
    d_seq_rc = (char**) malloc(NUM_DEVICES*sizeof(char*));
    
    d_index_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_pos_table = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));

    d_seed_offsets = (uint64_t**) malloc(NUM_DEVICES*sizeof(uint64_t*));

    d_hit_num_array = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_hit_num_vec.reserve(NUM_DEVICES);

    d_done = (uint32_t**) malloc(NUM_DEVICES*sizeof(uint32_t*));
    d_done_vec.reserve(NUM_DEVICES);

    d_hsp = (segment**) malloc(NUM_DEVICES*sizeof(segment*));
    d_hsp_vec.reserve(NUM_DEVICES);

    d_hsp_reduced = (segment**) malloc(NUM_DEVICES*sizeof(segment*));
    d_hsp_reduced_vec.reserve(NUM_DEVICES);

    segment zeroHsp;
    zeroHsp.ref_start = 0;
    zeroHsp.query_start = 0;
    zeroHsp.len = 0;
    zeroHsp.score = 0;

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "InitializeProcessor");

        check_cuda_malloc((void**)&d_sub_mat[g], NUC2*sizeof(int), "sub_mat"); 

        check_cuda_memcpy((void*)d_sub_mat[g], (void*)sub_mat, NUC2*sizeof(int), hipMemcpyHostToDevice, "sub_mat");

        check_cuda_malloc((void**)&d_seed_offsets[g], MAX_SEEDS*sizeof(uint64_t), "seed_offsets");

        d_hit_num_vec.emplace_back(MAX_SEEDS, 0);
        d_hit_num_array[g] = thrust::raw_pointer_cast(d_hit_num_vec.at(g).data());

        d_done_vec.emplace_back(MAX_HITS, 0);
        d_done[g] = thrust::raw_pointer_cast(d_done_vec.at(g).data());

        d_hsp_vec.emplace_back(MAX_HITS, zeroHsp);
        d_hsp[g] = thrust::raw_pointer_cast(d_hsp_vec.at(g).data());

        d_hsp_reduced_vec.emplace_back(MAX_HITS, zeroHsp);
        d_hsp_reduced[g] = thrust::raw_pointer_cast(d_hsp_reduced_vec.at(g).data());

        available_gpus.push_back(g);
    }
    
    return NUM_DEVICES;
}

void InclusivePrefixScan (uint32_t* data, uint32_t len) {
    int g;
    
    {
        std::unique_lock<std::mutex> locker(mu);
        if (available_gpus.empty()) {
            cv.wait(locker, [](){return !available_gpus.empty();});
        }
        g = available_gpus.back();
        available_gpus.pop_back();
        locker.unlock();

        check_cuda_setDevice(g, "InclusivePrefixScan");
    }

    thrust::inclusive_scan(thrust::host, data, data + len, data); 

    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(g);
        locker.unlock();
        cv.notify_one();
    }
}

void SendSeedPosTable (uint32_t* index_table, uint32_t index_table_size, uint32_t* pos_table, uint32_t num_index){

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "SendSeedPosTable");

        check_cuda_malloc((void**)&d_index_table[g], index_table_size*sizeof(uint32_t), "index_table"); 

        check_cuda_memcpy((void*)d_index_table[g], (void*)index_table, index_table_size*sizeof(uint32_t), hipMemcpyHostToDevice, "index_table");

        check_cuda_malloc((void**)&d_pos_table[g], num_index*sizeof(uint32_t), "pos_table"); 

        check_cuda_memcpy((void*)d_pos_table[g], (void*)pos_table, num_index*sizeof(uint32_t), hipMemcpyHostToDevice, "pos_table");
    }
}

void SendRefWriteRequest (size_t start_addr, uint32_t len){

    seq_len = len;
    
    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "SendRefWriteRequest");

        char* d_seq_tmp;
        check_cuda_malloc((void**)&d_seq_tmp, len*sizeof(char), "d_seq_tmp"); 

        check_cuda_memcpy((void*)d_seq_tmp, (void*)(seq_DRAM->buffer + start_addr), len*sizeof(char), hipMemcpyHostToDevice, "seq");

        check_cuda_malloc((void**)&d_seq[g], len*sizeof(char), "seq"); 
        check_cuda_malloc((void**)&d_seq_rc[g], len*sizeof(char), "seq_rc"); 

        compress_string_rev_comp <<<MAX_BLOCKS, MAX_THREADS>>> (len, d_seq_tmp, d_seq[g], d_seq_rc[g]);

        check_cuda_free((void*)d_seq_tmp, "d_seq_tmp");
    }
}

void clearRef(){

    for(int g = 0; g < NUM_DEVICES; g++){

        check_cuda_setDevice(g, "clearRef");

        check_cuda_free((void*)d_seq[g], "d_seq");
        check_cuda_free((void*)d_seq_rc[g], "d_seq_rc");
        check_cuda_free((void*)d_index_table[g], "d_index_table");
        check_cuda_free((void*)d_pos_table[g], "d_pos_table");
    }
}

void ShutdownProcessor(){

    d_done_vec.clear();
    d_hit_num_vec.clear();
    d_hsp_vec.clear();
    d_hsp_reduced_vec.clear();

    hipDeviceReset();
}

InitializeProcessor_ptr g_InitializeProcessor = InitializeProcessor;
InclusivePrefixScan_ptr g_InclusivePrefixScan = InclusivePrefixScan;
SendSeedPosTable_ptr g_SendSeedPosTable = SendSeedPosTable;
SendRefWriteRequest_ptr g_SendRefWriteRequest = SendRefWriteRequest;
SeedAndFilter_ptr g_SeedAndFilter = SeedAndFilter;
clearRef_ptr g_clearRef = clearRef;
ShutdownProcessor_ptr g_ShutdownProcessor = ShutdownProcessor;
